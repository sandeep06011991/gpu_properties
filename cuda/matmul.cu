#include<iostream>
#include "hip/hip_runtime.h"
#include "assert.h"
#include <chrono>

#define N (1<<11)

void local_execute(int *c, int *local_a, int *local_b){
    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            int s =0;
            for(int k=0;k<N;k++){
                c[i*N+j] = s  + local_a[i*N + k]*local_b[j + k*N];
            }
        }
    }
}
//
//int verify(int *c){
//    int error = 0;
//    for(int i=0;i<size;i++){
//        error = error + abs(4-c[i]);
//    }
//    return error;
//}
//
void check_error(hipError_t e){
    assert(e == hipSuccess);

}
//
__global__ void matmul_gpu(int *a,int *b,int *c){
    int i = blockIdx.x;
    for(int j=0;j<N;j++){
        int s =0;
        for(int k=0;k<N;k++){
            c[i*N+j] = s  + a[i*N + k]*b[j + k*N];
        }
    }
}
//
void gpu_execute(int *local_a, int* local_b, int *local_c){
    int *a,*b,*c;
    check_error(hipMalloc(&a, N * N * sizeof(int)));
    check_error(hipMalloc(&b, N * N * sizeof(int)));
    check_error(hipMalloc(&c, N * N * sizeof(int)));
    check_error(hipMemcpy(a,local_a,N * N * sizeof(int),hipMemcpyHostToDevice));
    check_error(hipMemcpy(b,local_b,N * N * sizeof(int),hipMemcpyHostToDevice));
    matmul_gpu<<<N,1>>>(a,b,c);
    check_error(hipMemcpy(local_c,c,N * N * sizeof(int),hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    hipFree(a);
    hipFree(b);
    hipFree(c);

}
/*
 * Current Speed UP:
 * GPU Run time 3904ms
 * Local run time 80398ms
*/
int main(){

    int * local_a = (int *)malloc(sizeof(int) * N * N);
    int * local_b = (int *)malloc(sizeof(int) * N * N);
    int * c = (int *)malloc(sizeof(int) * N * N);

    for(int i=0;i<N;i++){
        local_a[i]= 1;
        local_b[i]= 1;
    }
    std::cout << "Matrix Size" << ((N * N * 4)/(1<<20))  <<"MB\n";
    auto start_time = std::chrono::high_resolution_clock::now();
    gpu_execute(c,local_a,local_b);
    auto end_time = std::chrono::high_resolution_clock::now();
    std::cout << "GPU Run time " << (end_time - start_time)/std::chrono::milliseconds(1) <<"ms \n";
    start_time = std::chrono::high_resolution_clock::now();
    local_execute(c,local_a,local_b);
    end_time = std::chrono::high_resolution_clock::now();
    std::cout << "Local run time " << (end_time - start_time)/std::chrono::milliseconds(1) <<"ms \n";
    free(local_a);
    free(local_b);
    free(c);
    //    gpu_execute(local_a,local_b,c);
//    std::cout << "Max Error" << verify(c) <<"\n";
}
