
#include "common.cuh"




void pinnedTransfer(size_t N){
    int * local_a;
    check_error(hipHostAlloc(&local_a,sizeof(int) * N, hipHostMallocDefault));
    int * a;
    check_error(hipMalloc(&a, N * sizeof(int)));
    auto start_time = std::chrono::high_resolution_clock::now();
    check_error(hipMemcpy(a,local_a,N *  sizeof(int),hipMemcpyHostToDevice));
    hipDeviceSynchronize();
    auto end_time = std::chrono::high_resolution_clock::now();
    std::cout << (end_time - start_time)/std::chrono::milliseconds(1) <<"ms \n";
    hipFree(a);
    hipHostFree(local_a);
}

void transfer(size_t N){
    int * local_a =(int *)malloc(sizeof(int) * N);
    int * a;
    check_error(hipMalloc(&a, N * sizeof(int)));
    auto start_time = std::chrono::high_resolution_clock::now();
    check_error(hipMemcpy(a,local_a,N *  sizeof(int),hipMemcpyHostToDevice));
    hipDeviceSynchronize();
    auto end_time = std::chrono::high_resolution_clock::now();
    std::cout << (end_time - start_time)/std::chrono::milliseconds(1) <<"ms \n";
    hipFree(a);
    free(local_a);
}

int main(){
    for(int i=15;i<30;i++){
//      bytes
        size_t bytes = (1<<i) * sizeof(int);
//        cout << bytes << " " << (bytes/(1<<10)) << " ";
        if(bytes <1024){
            cout << bytes <<"bytes ";
        }
        if((bytes >= 1024) && (bytes < 1024*1024)){
            cout << (bytes / (1<<10))<<"KB ";
        }
        if((bytes < (1<<30)) && (bytes>=(1<<20))){
            cout << bytes / (1<<20)<<"MB ";
        }
        if(bytes >= (1<<30)){
            cout << bytes / (1<<30) << "GB ";
        }
        cout <<" | ";
        transfer(1<<i);
//        pinnedTransfer(1<<i);
    }
}