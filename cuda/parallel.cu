#include "hip/hip_runtime.h"

#include "common.cuh"



__global__ void timetakingfunction(){
    int s = 0;
    for(int i=0;i<10000000;i++){
        s=s*19;
    }
}


int main(){
//    for(int i=1;i<100000;i=i*10){
//        auto start_time = std::chrono::high_resolution_clock::now();
//        timetakingfunction<<<i,1>>>();
//        hipDeviceSynchronize();
//        auto end_time = std::chrono::high_resolution_clock::now();
//        cout << "Thread " << i <<  ":" << (end_time - start_time)/std::chrono::microseconds(1) <<"ms \n";
//    }
    for(int i=10;i<100000;i=i*10){
        auto start_time = std::chrono::high_resolution_clock::now();
        timetakingfunction<<<(i/32)+1,5000>>>();
        hipDeviceSynchronize();
        auto end_time = std::chrono::high_resolution_clock::now();
        cout << "Thread " << i <<  ":" << (end_time - start_time)/std::chrono::microseconds(1) <<"ms \n";
    }
}