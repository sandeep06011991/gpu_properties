#include "hip/hip_runtime.h"

#include "common.cuh"



__global__ void timetakingfunction(int *a){
    int s = 0;
    for(int i=0;i<1000;i++){
        s=s*19;
    }
    a[threadIdx.x] = s;
}

/* Code framework to test different kinds of kernel launches
 * with varying grid sizes threads and blocks. */
int main(){
    int *a;
    check_error(hipMalloc(&a, 10000000 * sizeof(int)));
    for(int i=1;i<10;i=i+10){
        auto start_time = std::chrono::high_resolution_clock::now();
        timetakingfunction<<<100,128>>>(a);
        check_error( hipPeekAtLastError() );
        hipDeviceSynchronize();
        auto end_time = std::chrono::high_resolution_clock::now();
        cout << "Thread " << i <<  ":" << (end_time - start_time)/std::chrono::microseconds(1) <<"ms \n";
    }

}