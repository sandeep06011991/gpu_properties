
# include <hip/hip_runtime.h>
# include "hipblas.h"
#include <iostream>
#define M (1<<11)
#define N (1<<11)
#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))
using namespace std;

int main(){

	hipError_t cudaStat;
	hipblasStatus_t stat;    
	hipblasHandle_t handle;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	int i, j;
	float* devPtrA;
	float* devPtrC;
	float* a = 0;    
	float * res;
	a = (float *)malloc (M * N * sizeof (*a));
	res = (float *)malloc(M * N * sizeof(*a));
	if (!a) {
		printf ("host memory allocation failed");
		return EXIT_FAILURE;   
       	}
	for (j = 1; j <= N; j++) {
		for (i = 1; i <= M; i++) {
	    		a[IDX2F(i,j,M)] = (float)(3);
		}
    	}   
       	cudaStat = hipMalloc ((void**)&devPtrA, M*N*sizeof(*a));
	cudaStat = hipMalloc((void **)&devPtrC, M*N*sizeof(float));
	if (cudaStat != hipSuccess) {
		printf ("device memory allocation failed");
		return EXIT_FAILURE;   
       	}   
       	stat = hipblasCreate(&handle);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf ("CUBLAS initialization failed\n");
		return EXIT_FAILURE;   
       	}    
	hipEventRecord(start);
	stat = hipblasSetMatrix (M, N, sizeof(*a), a, M, devPtrA, M);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf ("data download failed");
		hipFree (devPtrA);
		hipblasDestroy(handle);
		return EXIT_FAILURE;
    	}
        
	const float alpha = 1.0f;
	const float beta = 0.0f;
	stat =  hipblasSgemm(handle,
		HIPBLAS_OP_N, HIPBLAS_OP_N, 
		M , N, N,
		&alpha, 
		devPtrA, M,
		devPtrA, N,
		&beta,
		devPtrC, M);

	if (stat != HIPBLAS_STATUS_SUCCESS){
		printf("mat mul fialed\n");
	}

	stat = hipblasGetMatrix( M, N, sizeof(float), devPtrC, M, res, M);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

//	Same Matrix too 23ms .
	cout << "Total time" << milliseconds <<"ms \n";
//	for(int j=1;j <=N; j++){
//		for(int i=1;i<=M;i++){
//			cout << res[IDX2F(i,j,M)] <<" ";
//		}
//		cout <<"\n";
//	}

}
