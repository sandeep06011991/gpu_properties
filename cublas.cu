
# include <hip/hip_runtime.h>
# include "hipblas.h"
#include <iostream>
#define M 5
#define N 5
#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))
using namespace std;

int main(){

	hipError_t cudaStat;
	hipblasStatus_t stat;    
	hipblasHandle_t handle;
	int i, j;
	float* devPtrA;
	float* devPtrC;
	float* a = 0;    
	float * res;
	a = (float *)malloc (M * N * sizeof (*a));
	res = (float *)malloc(M * N * sizeof(*a));
	if (!a) {
		printf ("host memory allocation failed");
		return EXIT_FAILURE;   
       	}
	for (j = 1; j <= N; j++) {
		for (i = 1; i <= M; i++) {
	    		a[IDX2F(i,j,M)] = (float)(3);
		}
    	}   
       	cudaStat = hipMalloc ((void**)&devPtrA, M*N*sizeof(*a));
	cudaStat = hipMalloc((void **)&devPtrC, M*N*sizeof(float));
	if (cudaStat != hipSuccess) {
		printf ("device memory allocation failed");
		return EXIT_FAILURE;   
       	}   
       	stat = hipblasCreate(&handle);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf ("CUBLAS initialization failed\n");
		return EXIT_FAILURE;   
       	}    
	stat = hipblasSetMatrix (M, N, sizeof(*a), a, M, devPtrA, M);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf ("data download failed");
		hipFree (devPtrA);
		hipblasDestroy(handle);
		return EXIT_FAILURE;
    	}
        
	const float alpha = 1.0f;
	const float beta = 0.0f;
	stat =  hipblasSgemm(handle,
		HIPBLAS_OP_N, HIPBLAS_OP_N, 
		M , N, N,
		&alpha, 
		devPtrA, M,
		devPtrA, N,
		&beta,
		devPtrC, M);

	if (stat != HIPBLAS_STATUS_SUCCESS){
		printf("mat mul fialed\n");
	}

	stat = hipblasGetMatrix( M, N, sizeof(float), devPtrC, M, res, M);

	for(int j=1;j <=N; j++){
		for(int i=1;i<=M;i++){
			cout << res[IDX2F(i,j,M)] <<" ";
		}
		cout <<"\n";
	}
	std::cout << "Hello World\n";

}
