#include "hip/hip_runtime.h"
#include<iostream>
#include "hip/hip_runtime.h"
#include "assert.h"

#define N (1<<15)

void local_execute(int *c, int *local_a, int *local_b){
    for(int i=0;i<size;i++){
        int s =0;
        for(int j=0;j<size;j++){
            c[i] = local_a[i] + local_b[i];
        }
    }
}
//
//int verify(int *c){
//    int error = 0;
//    for(int i=0;i<size;i++){
//        error = error + abs(4-c[i]);
//    }
//    return error;
//}
//
//void check_error(hipError_t e){
//    assert(e == hipSuccess);
//
//}
//
//__global__ void add(int *a,int *b,int *c){
//    int x = blockIdx.x;
//    c[x] = a[x] + b[x];
//}
//
//void gpu_execute(int *local_a, int* local_b, int *local_c){
//    int *a,*b,*c;
//    check_error(hipMalloc(&a, size *sizeof(int)));
//    check_error(hipMalloc(&b, size *sizeof(int)));
//    check_error(hipMalloc(&c, size *sizeof(int)));
//    check_error(hipMemcpy(a,local_a,size * sizeof(int),hipMemcpyHostToDevice));
//    check_error(hipMemcpy(b,local_b,size * sizeof(int),hipMemcpyHostToDevice));
//    add<<<size,1>>>(a,b,c);
//    check_error(hipMemcpy(local_c,c,size * sizeof(int),hipMemcpyDeviceToHost));
//    hipDeviceSynchronize();
//    hipFree(a);
//    hipFree(b);
//    hipFree(c);
//
//}

int main(){

    int * local_a = (int *)malloc(sizeof(int) * N * N);
    int * local_b = (int *)malloc(sizeof(int) * N * N);
    int * c = (int *)malloc(sizeof(int) * N * N);

    for(int i=0;i<N;i++){
        local_a[i]= 1;
        local_b[i]= 1;
    }

    local_execute(c,local_a,local_b);
//    gpu_execute(local_a,local_b,c);
//    std::cout << "Max Error" << verify(c) <<"\n";
}
